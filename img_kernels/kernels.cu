#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
extern "C" {
#include "defs.h"
#include "ppm.h"
}

/* There needs to be some way to map the block idx, thd indx to a chunk of the picture
 * 
 * Under what circumstances would the kernel need to cover more than one pixel?
 *    num_pixels / (blocks * thds) > 1
 * we know num_pixels = dim * dim;
 * we know thds = blockDim.x
 * we know blocks = gridDim.x
 * 
 * So what happens with the following scenario?
 *  dim = 18
 *  thds = 32
 *  blocks = 8
 * 
 *  num_pixels = 18*18 = 324
 *  324 / (32 * 8) = ~1.26
 *  so... each thread covers 1.26 pixels? Options...
 *      Launch another kernel for the remainder
 *      Kernel takes care of ceil(1.26) pixels per thread
 *        But then I've launched 256 thds, but will only be using 162 of them.
 *        Maybe that's just a side effect of the images I choose
 *      Only some threads are taking 2 pixels per thread while others are taking 1
 *        This gets around dead threads, but still will have some kind of if block
 *        that is going to leave some parts of the grid idle for some instructions.
 *        Is this an improvement over the ceil(pixel/thread) strategy? ... yes.
 *          You are using all threads to execute the kernel and the only time a part of the
 *          grid is idle is within an if block statement. Contrast with the ceil strategy
 *          and get that same amount of grid entirely dead.
 * 
 * How then to decide which threads are taking the lower count and which threads are taking
 * the higher count?
 *  Loose algorithm for this:
 *    Choose n threads to execute ceil(pixel/thread) such that: 
 *      (num_pixels - n) / floor(pixel/thread) = 1 or very near 1
 *  Need to optimize this system of equations:
 *    2x + y >= num_pixels
 *    2x + y <= threads
 *
 *  ^^^ Seems a bit too complex ^^^
 *  why not just use mod?
 *    n = num_pixels % threads
 *    m = num_pixels - n
 *    n will be the threads that execute the ceil(pixel/thread) version
 *    m will be the threads that execute the floor(pixel/thread) version
 *    
 *  So now how to map a thread to its proper index, considering it now needs to know
 *  how many pixels the threads prior to it have taken?
 * 
 *  ^^^ Still too complex... ^^^
 *  Doesn't the number of threads per block you pick force the number of blocks you pick?
 *  And vise versa... pick 
 *  
 * .---------------.
 * |_|_|_|_|_|_|_|_|
 * |_|_|_|_|_|_|_|_|
 * |_|_|_|_|_|_|_|_|
 * |_|_|_|_|_|_|_|_|
 * |_|_|_|_|_|_|_|_|
 * |_|_|_|_|_|_|_|_|
 * |_|_|_|_|_|_|_|_|
 * | | | | | | | | |
 * .---------------.

 * 30 May 2017
 * 
 * So the question that still is floating around after sleeping on this is whether a kernel
 * is better off doing a 1 pixel operation for every thread while leaving big fractions of
 * the kernel completely idle, or if a kernel is better off mapping work to every thread
 * available. That second option forces the kernel to launch some threads that
 * don't execute every instruction...
 */  
__global__
void complex_kernel(int dim, rgb_pixel* src, rgb_pixel* dest) {
  int c_stride = blockDim.x;
  int r_stride = gridDim.y;
  int gridWidth = ceil((float)dim / (float)gridDim.x);
  int j_anchor = blockIdx.x * gridWidth;

  int i, j;
  for(i=blockIdx.y; i < dim; i+=r_stride) {
    for(j = j_anchor + threadIdx.x; j < (j_anchor + gridWidth) && j < dim; j+=c_stride) {
      rgb_pixel px = src[(i*dim) + j];
      px.r = px.g = px.b = ((int)px.r + (int)px.g + (int)px.b) / 3;

      int dest_r, dest_c;
      dest_r = (dim - j - 1);
      dest_c = (dim - i - 1);
      dest[(dest_r * dim) + dest_c] = px;
    }
  }
}

__global__
void complex_kernel_chunk(int dim, rgb_pixel* src, rgb_pixel* dest) {
  int c_chunk_size = ceil((float)dim / (float)(blockDim.x * gridDim.x));;
  int r_chunk_size = ceil((float)dim / (float)gridDim.y);
  int grid_width = ceil((float)dim / (float)gridDim.x);
  int c_anchor = (blockIdx.x * grid_width) + (threadIdx.x * c_chunk_size);
  int r_anchor = (blockIdx.y * r_chunk_size);  

  int i, j;
  for(i=r_anchor; i < r_anchor + r_chunk_size && i < dim; i++) {
    for(j = c_anchor; j < (c_anchor + c_chunk_size) && j < dim; j++) {
      rgb_pixel px = src[(i*dim) + j];
      px.r = px.g = px.b = ((int)px.r + (int)px.g + (int)px.b) / 3;

      int dest_r, dest_c;
      dest_r = (dim - j - 1);
      dest_c = (dim - i - 1);
      dest[(dest_r * dim) + dest_c] = px;
    }
  }
}

__host__
void launch_complex_kernel(int gridX, int gridY, int block, int dim, rgb_pixel* d_src, rgb_pixel** h_dest) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  printf("Launching complex kernel...\n");
  printf("Blocks: x: %d, y: %d \t Threads: %d\n", gridX, gridY, block);
  rgb_pixel* d_dest;
  hipMalloc((void**) &d_dest, sizeof(rgb_pixel) * dim * dim); 
 
  dim3 grd(gridX, gridY);
  dim3 blk(block);

  hipEventRecord(start);
  complex_kernel_chunk<<<grd, blk>>>(dim, d_src, d_dest);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  printf("Kernel execution time: %f\n", ms);

  *h_dest = (rgb_pixel*)malloc(sizeof(rgb_pixel) * dim * dim);
  hipMemcpy(*h_dest, d_dest, sizeof(rgb_pixel) * dim * dim, hipMemcpyDeviceToHost);

  hipFree(d_src); hipFree(d_dest);
}

__global__
void motion_kernel(int dim, rgb_pixel* src, rgb_pixel* dest) {
  int c_stride = blockDim.x;
  int r_stride = gridDim.y;
  int gridWidth = ceil((float)dim / (float)gridDim.x);
  int j_anchor = blockIdx.x * gridWidth;

  int i, j;
  for(i=blockIdx.y; i < dim; i+=r_stride) {
    for(j = j_anchor + threadIdx.x; j < (j_anchor + gridWidth) && j < dim; j+=c_stride) {
      int ii, jj;
      int r, g, b;
      r = g = b = 0;

      int num_neighbors = 0;

      for(ii=0; ii < 3; ii++) {
        for(jj=0; jj < 3; jj++) {
          if((i+ii < dim) && (j + jj < dim)) {
            num_neighbors++;
            rgb_pixel px_n = src[((i+ii)*dim) + jj + j];
            r += (int)px_n.r;
            g += (int)px_n.g;
            b += (int)px_n.b; 
          }
        }
      }
      
      dest[(i * dim) + j].r = r / num_neighbors;
      dest[(i * dim) + j].g = g / num_neighbors;
      dest[(i * dim) + j].b = b / num_neighbors;
    }
  }
}



__global__
void motion_kernel_halo(int dim, rgb_pixel* src, rgb_pixel* dest) {
  int c_stride = blockDim.x;
  int r_stride = gridDim.y;
  int gridWidth = ceil((float)dim / (float)gridDim.x);
  int j_anchor = blockIdx.x * gridWidth;
  
  extern __shared__ rgb_pixel shared[];
    
  int i, j;
  for(i=blockIdx.y; i < dim; i+=r_stride) {
    for(j = j_anchor + threadIdx.x; j < (j_anchor + gridWidth) && j < dim; j+=c_stride) {
      shared[threadIdx.x] = src[i * dim + j];
      if(i + 1 < dim){
        shared[threadIdx.x + c_stride] = src[(i + 1) * dim + j];
      }
      if(i + 2 < dim){
        shared[threadIdx.x + (2 * c_stride)] = src[(i+2) * dim + j];
      }

      __syncthreads();

      int ii, jj;
      int r, g, b;
      r = g = b = 0;

      int num_neighbors = 0;

      for(ii=0; ii < 3; ii++) {
        for(jj=0; jj < 3; jj++) {
          if((i+ii < dim) && (j + jj < dim)) {
            num_neighbors++;
            rgb_pixel px_n;
            if(threadIdx.x + jj < c_stride) {
              px_n = shared[ii * c_stride + threadIdx.x + jj];
            } else {
              px_n = src[((i+ii)*dim) + jj + j];
            }
            r += (int)px_n.r;
            g += (int)px_n.g;
            b += (int)px_n.b; 
          }
        }
      }
      
      dest[(i * dim) + j].r = r / num_neighbors;
      dest[(i * dim) + j].g = g / num_neighbors;
      dest[(i * dim) + j].b = b / num_neighbors;
    }
  }
}

__host__
void launch_motion_kernel(int gridX, int gridY, int block, int dim, rgb_pixel* d_src, rgb_pixel** h_dest) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  printf("Launching motion kernel...\n");
  printf("Blocks: x: %d, y: %d \t Threads: %d\n", gridX, gridY, block);
  rgb_pixel* d_dest;
  hipMalloc((void**) &d_dest, sizeof(rgb_pixel) * dim * dim); 
 
  dim3 grd(gridX, gridY);
  dim3 blk(block);

  hipEventRecord(start);
  motion_kernel<<<grd, blk>>>(dim, d_src, d_dest);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  printf("Kernel execution time: %f\n", ms);

  *h_dest = (rgb_pixel*)malloc(sizeof(rgb_pixel) * dim * dim);
  hipMemcpy(*h_dest, d_dest, sizeof(rgb_pixel) * dim * dim, hipMemcpyDeviceToHost);

  hipFree(d_src); hipFree(d_dest);
}

